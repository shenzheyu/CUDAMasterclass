#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void occupany_test(int *results) {
	int gid = blockDim.x * blockIdx.x + threadIdx.x;

	int x1 = 1;
	int x2 = 2;
	int x3 = 3;
	int x4 = 4;
	int x5 = 5;
	int x6 = 6;
	int x7 = 7;
	int x8 = 8;

	results[gid] = x1 + x2 + x3 + x4 + x5 + x6 + x7 +x8;
}

int main()
{
	int iDev = 0;
	hipDeviceProp_t iProp;

	hipGetDeviceProperties(&iProp, iDev);
	printf("Max threads per SM : %d \n",
		iProp.maxThreadsPerMultiProcessor);

	return 0;
}