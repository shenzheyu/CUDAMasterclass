#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void print_details()
{
	printf("blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, gridDim.x : %d, gridDim.y :%d, gridDim.z :%d \n",
		blockIdx.x, blockIdx.y, blockIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
}

int main()
{
	dim3 block(4, 4, 4);
	dim3 grid(2, 2, 2);

	print_details << <grid, block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}
