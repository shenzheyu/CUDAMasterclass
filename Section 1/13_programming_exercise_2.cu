#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void unique_gid_calculation_3d_3d(int * data) {
    int tid = blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x;

	int num_threads_in_a_block = blockDim.x * blockDim.y * blockDim.z;
	int block_offset = blockIdx.x * num_threads_in_a_block;

	int num_threads_in_a_row = num_threads_in_a_block * gridDim.x;
	int row_offset = num_threads_in_a_row * blockIdx.y;

    int num_threads_in_a_column = num_threads_in_a_row * gridDim.y;
    int column_offset = num_threads_in_a_column * blockIdx.z;

	int gid = tid + block_offset + row_offset + column_offset;

	printf("blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, threadIdx.x : %d, gid : %d - data : %d \n",
		blockIdx.x, blockIdx.y, blockIdx.z, tid, gid, data[gid]);
}

int main() {
	int size = 64;
	int byte_size = size * sizeof(int);

	int * h_input;
	h_input = (int*)malloc(byte_size);

	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; i++)
	{
		h_input[i] = (int)(rand() & 0xff);
	}

	int * d_input;
	hipMalloc((void**)&d_input, byte_size);

	hipMemcpy(d_input,h_input,byte_size,hipMemcpyHostToDevice);

	dim3 block(2, 2, 2);
	dim3 grid(2, 2, 2);

	unique_gid_calculation_3d_3d<<<grid, block>>>(d_input);
	hipDeviceSynchronize();

	hipFree(d_input);
	free(h_input);

	hipDeviceReset();
	return 0;
}